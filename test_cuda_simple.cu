// Simple CUDA compilation test
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void testKernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %d\n", idx);
}

int main() {
    std::cout << "Testing CUDA compilation..." << std::endl;
    
    // Check CUDA device
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    
    std::cout << "Found " << deviceCount << " CUDA devices" << std::endl;
    
    // Launch simple kernel
    testKernel<<<1, 4>>>();
    hipDeviceSynchronize();
    
    error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "Kernel Error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    
    std::cout << "CUDA test completed successfully!" << std::endl;
    return 0;
}
