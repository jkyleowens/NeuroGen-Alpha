// CUDA Compatibility and type trait fixes
#include <NeuroGen/cuda/CudaCompatibility.h>
#include <NeuroGen/cuda/CudaUtils.h>

#include "../../include/NeuroGen/cuda/KernelLaunchWrappers.cuh"
#include "../../include/NeuroGen/cuda/NeuronUpdateKernel.cuh"
#include "../../include/NeuroGen/cuda/NeuronSpikingKernels.cuh"
#include "../../include/NeuroGen/cuda/SynapseInputKernel.cuh"
#include "../../include/NeuroGen/cuda/STDPKernel.cuh"
#include "../../include/NeuroGen/cuda/RandomStateInit.cuh"
#include "../../include/NeuroGen/cuda/GridBlockUtils.cuh"
#include "../../include/NeuroGen/GPUNeuralStructures.h"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

extern "C" void launchUpdateNeuronVoltages(GPUNeuronState* neurons, float* I_leak, float* Cm, float dt, int N) {
    dim3 block = makeBlock();
    dim3 grid = makeGrid(N);
    updateNeuronVoltages<<<grid, block>>>(neurons, I_leak, Cm, dt, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        // Handle error appropriately
    }
    hipDeviceSynchronize();
}

void launchNeuronUpdateKernel(GPUNeuronState* neurons, float dt, int N) {
    dim3 block = makeSafeBlock(256);
    dim3 grid = makeSafeGrid(N, 256);
    rk4NeuronUpdateKernel<<<grid, block>>>(neurons, dt, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        // Handle error appropriately
    }
    hipDeviceSynchronize();
}

extern "C" void launchSynapseInputKernel(GPUSynapse* d_synapses, GPUNeuronState* d_neurons, int num_synapses) {
    dim3 block = makeBlock();
    dim3 grid = makeGrid(num_synapses);
    synapseInputKernel<<<grid, block>>>(d_synapses, d_neurons, num_synapses);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        // Handle error appropriately
    }
    hipDeviceSynchronize();
}

void launchRK4NeuronUpdateKernel(GPUNeuronState* neurons, int N, float dt) {
    dim3 block = makeSafeBlock(256);
    dim3 grid = makeSafeGrid(N, 256);
    rk4NeuronUpdateKernel<<<grid, block>>>(neurons, dt, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        // Handle error appropriately
    }
    hipDeviceSynchronize();
}

void launchSpikeDetectionKernel(GPUNeuronState* neurons, GPUSpikeEvent* spikes, float threshold,
                                int* spike_count, int num_neurons, float current_time) {
    dim3 block = makeSafeBlock(256);
    dim3 grid = makeSafeGrid(num_neurons, 256);
    detectSpikes<<<grid, block>>>(neurons, spikes, threshold, spike_count, num_neurons, current_time);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        // Handle error appropriately
    }
    hipDeviceSynchronize();
}