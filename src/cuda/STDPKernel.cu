#include "hip/hip_runtime.h"
#include "../../include/NeuroGen/cuda/STDPKernel.cuh"
#include "../../include/NeuroGen/GPUNeuralStructures.h"
#include "../../include/NeuroGen/cuda/GridBlockUtils.cuh"
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void stdpUpdateKernel(GPUSynapse* synapses, const GPUNeuronState* neurons,
                                int num_synapses, float A_plus, float A_minus,
                                float tau_plus, float tau_minus, float current_time,
                                float min_weight, float max_weight, float reward_signal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    // Skip inactive synapses
    if (synapse.active == 0) return;
    
    int pre_idx = synapse.pre_neuron_idx;
    int post_idx = synapse.post_neuron_idx;
    
    // Get spike times
    float t_pre = neurons[pre_idx].last_spike_time;
    float t_post = neurons[post_idx].last_spike_time;
    
    // Skip if no recent spikes
    if (t_pre < 0.0f || t_post < 0.0f) return;
    
    // Calculate time differences
    float dt_pre_post = t_post - t_pre;
    
    // Apply STDP rule
    float dw = 0.0f;
    
    // LTP: post after pre (causal)
    if (dt_pre_post > 0.0f && dt_pre_post < 50.0f) {
        dw = A_plus * expf(-dt_pre_post / tau_plus);
    }
    // LTD: pre after post (acausal)
    else if (dt_pre_post < 0.0f && dt_pre_post > -50.0f) {
        dw = -A_minus * expf(dt_pre_post / tau_minus);
    }
    
    // Apply reward modulation
    dw *= (1.0f + reward_signal);
    
    // Update weight
    if (dw != 0.0f) {
        synapse.weight += dw;
        
        // Clamp weight to valid range
        if (synapse.weight < min_weight) synapse.weight = min_weight;
        if (synapse.weight > max_weight) synapse.weight = max_weight;
        
        // Update activity metric and last potentiation time
        if (dw > 0.0f) {
            synapse.last_potentiation = current_time;
        }
    }
}

void launchSTDPUpdateKernel(GPUSynapse* d_synapses, const GPUNeuronState* d_neurons,
                           int num_synapses, float A_plus, float A_minus,
                           float tau_plus, float tau_minus, float current_time,
                           float min_weight, float max_weight, float reward_signal) {
    dim3 block = makeBlock();
    dim3 grid = makeGrid(num_synapses);
    
    stdpUpdateKernel<<<grid, block>>>(d_synapses, d_neurons, num_synapses,
                                     A_plus, A_minus, tau_plus, tau_minus,
                                     current_time, min_weight, max_weight,
                                     reward_signal);
    
    hipDeviceSynchronize();
}