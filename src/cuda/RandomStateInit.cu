#include "hip/hip_runtime.h"
// RandomStateInit.cu - Kernel implementation only
#include "../../include/NeuroGen/cuda/RandomStateInit.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

__global__ void initRandomStates(hiprandState* states, int num_states, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= num_states) return;
    hiprand_init(seed, idx, 0, &states[idx]);
}


__global__ void initializeRandomStates(hiprandState* states, unsigned long seed, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        // Each thread gets same seed, different sequence number
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}