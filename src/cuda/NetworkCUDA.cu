#include "hip/hip_runtime.h"
#include "NetworkCUDA.cuh"
#include "CudaUtils.cuh"
#include "NetworkConfig.h"
#include "GPUNeuralStructures.h"
#include "STDPKernel.cuh"
#include "KernelLaunchWrappers.cuh"
#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <memory>
#include <chrono>

// Global network state
static NetworkConfig g_config;
static GPUNeuronState* d_neurons = nullptr;
static GPUSynapse* d_synapses = nullptr;
static float* d_input_buffer = nullptr;
static float* d_output_buffer = nullptr;
static float* d_reward_buffer = nullptr;
static GPUSpikeEvent* d_spike_events = nullptr;
static int* d_spike_count = nullptr;
static hiprandState* d_rng_states = nullptr;

// Network topology tracking
static int total_neurons = 0;
static int total_synapses = 0;
static int input_start, input_end;
static int hidden_start, hidden_end;
static int output_start, output_end;

// Performance monitoring
struct NetworkStats {
    float avg_firing_rate = 0.0f;
    float total_spikes = 0.0f;
    float avg_weight = 0.0f;
    float reward_signal = 0.0f;
    int update_count = 0;
    
    void reset() {
        avg_firing_rate = 0.0f;
        total_spikes = 0.0f;
        avg_weight = 0.0f;
        reward_signal = 0.0f;
    }
};

static NetworkStats g_stats;
static float current_time = 0.0f;

// Forward declarations for CUDA kernels
__global__ void injectInputCurrentImproved(GPUNeuronState* neurons, const float* input_data, 
                                          int input_size, float current_time, float scale);
__global__ void extractOutputImproved(const GPUNeuronState* neurons, float* output_buffer,
                                     int output_size, float current_time);
__global__ void applyRewardModulationImproved(GPUNeuronState* neurons, int num_neurons, float reward);
__global__ void computeNetworkStatistics(const GPUNeuronState* neurons, const GPUSynapse* synapses,
                                        int num_neurons, int num_synapses, float* stats);
__global__ void resetSpikeFlags(GPUNeuronState* neurons, int num_neurons);

// Initialize the enhanced neural network
void initializeNetwork() {
    // Use trading-optimized configuration by default
    g_config = NetworkPresets::trading_optimized();
    
    if (!g_config.validate()) {
        throw std::runtime_error("Invalid network configuration");
    }
    
    g_config.print();
    printDeviceInfo();
    
    // Calculate network dimensions
    total_neurons = g_config.input_size + g_config.hidden_size + g_config.output_size;
    
    // Set layer boundaries
    input_start = 0;
    input_end = g_config.input_size;
    hidden_start = g_config.input_size;
    hidden_end = g_config.input_size + g_config.hidden_size;
    output_start = g_config.input_size + g_config.hidden_size;
    output_end = total_neurons;
    
    std::cout << "[CUDA] Initializing network with " << total_neurons << " neurons..." << std::endl;
    
    // Allocate GPU memory with error checking
    safeCudaMalloc(&d_neurons, total_neurons);
    safeCudaMalloc(&d_input_buffer, g_config.input_size);
    safeCudaMalloc(&d_output_buffer, g_config.output_size);
    safeCudaMalloc(&d_reward_buffer, 1);
    safeCudaMalloc(&d_spike_events, total_neurons * 10); // Buffer for multiple spikes
    safeCudaMalloc(&d_spike_count, 1);
    safeCudaMalloc(&d_rng_states, total_neurons);
    
    // Initialize neurons with proper HH resting state
    std::vector<GPUNeuronState> host_neurons(total_neurons);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> voltage_dist(-65.0f, 2.0f);
    std::uniform_real_distribution<float> gating_dist(0.0f, 0.02f);
    
    for (int i = 0; i < total_neurons; ++i) {
        auto& neuron = host_neurons[i];
        neuron.voltage = voltage_dist(gen);
        neuron.spiked = false;
        neuron.last_spike_time = -1000.0f;
        
        // Initialize HH gating variables near resting state
        neuron.m = 0.05f + gating_dist(gen);
        neuron.h = 0.60f + gating_dist(gen);
        neuron.n = 0.32f + gating_dist(gen);
        
        // Single compartment for now
        neuron.compartment_count = 1;
        neuron.voltages[0] = neuron.voltage;
        neuron.I_leak[0] = 0.0f;
        neuron.Cm[0] = 1.0f;
    }
    
    safeCudaMemcpy(d_neurons, host_neurons.data(), total_neurons, hipMemcpyHostToDevice);
    
    // Generate improved network topology
    std::vector<GPUSynapse> host_synapses;
    createNetworkTopology(host_synapses, gen);
    
    total_synapses = host_synapses.size();
    std::cout << "[CUDA] Created " << total_synapses << " synapses" << std::endl;
    
    // Copy synapses to GPU
    safeCudaMalloc(&d_synapses, total_synapses);
    safeCudaMemcpy(d_synapses, host_synapses.data(), total_synapses, hipMemcpyHostToDevice);
    
    // Initialize random states
    launchRandomStateInit(d_rng_states, total_neurons, rd());
    CUDA_CHECK_KERNEL();
    
    // Zero out buffers
    safeCudaMemset(d_input_buffer, 0, g_config.input_size);
    safeCudaMemset(d_output_buffer, 0, g_config.output_size);
    safeCudaMemset(d_spike_count, 0, 1);
    
    std::cout << "[CUDA] Network initialization complete!" << std::endl;
    g_stats.reset();
}

// Enhanced network topology creation
void createNetworkTopology(std::vector<GPUSynapse>& synapses, std::mt19937& gen) {
    std::uniform_real_distribution<float> weight_dist(0.0f, g_config.weight_init_std);
    std::uniform_real_distribution<float> delay_dist(g_config.delay_min, g_config.delay_max);
    std::uniform_real_distribution<float> prob_dist(0.0f, 1.0f);
    
    synapses.clear();
    synapses.reserve(total_neurons * 50); // Conservative estimate
    
    // Input to Hidden connections with cortical organization
    for (int pre = input_start; pre < input_end; ++pre) {
        for (int post = hidden_start; post < hidden_end; ++post) {
            if (prob_dist(gen) < g_config.input_hidden_prob) {
                GPUSynapse syn;
                syn.pre_neuron_idx = pre;
                syn.post_neuron_idx = post;
                syn.weight = weight_dist(gen) * (prob_dist(gen) < g_config.exc_ratio ? 1.0f : -1.0f);
                syn.delay = delay_dist(gen);
                syn.last_pre_spike_time = -1000.0f;
                syn.activity_metric = 0.0f;
                synapses.push_back(syn);
            }
        }
    }
    
    // Hidden layer recurrent connections (sparse)
    for (int pre = hidden_start; pre < hidden_end; ++pre) {
        for (int post = hidden_start; post < hidden_end; ++post) {
            if (pre != post && prob_dist(gen) < g_config.hidden_hidden_prob) {
                GPUSynapse syn;
                syn.pre_neuron_idx = pre;
                syn.post_neuron_idx = post;
                syn.weight = weight_dist(gen) * (prob_dist(gen) < g_config.exc_ratio ? 0.5f : -0.8f);
                syn.delay = delay_dist(gen);
                syn.last_pre_spike_time = -1000.0f;
                syn.activity_metric = 0.0f;
                synapses.push_back(syn);
            }
        }
    }
    
    // Hidden to Output connections (fully connected)
    for (int pre = hidden_start; pre < hidden_end; ++pre) {
        for (int post = output_start; post < output_end; ++post) {
            GPUSynapse syn;
            syn.pre_neuron_idx = pre;
            syn.post_neuron_idx = post;
            syn.weight = weight_dist(gen) * 0.3f; // Smaller initial weights for output
            syn.delay = delay_dist(gen);
            syn.last_pre_spike_time = -1000.0f;
            syn.activity_metric = 0.0f;
            synapses.push_back(syn);
        }
    }
}

// Enhanced forward pass with better dynamics
std::vector<float> forwardCUDA(const std::vector<float>& input, float reward_signal) {
    if (input.size() != g_config.input_size) {
        std::cerr << "[ERROR] Input size mismatch: expected " << g_config.input_size 
                  << ", got " << input.size() << std::endl;
        return std::vector<float>(g_config.output_size, 1.0f / g_config.output_size);
    }
    
    // Copy input to GPU
    safeCudaMemcpy(d_input_buffer, input.data(), g_config.input_size, hipMemcpyHostToDevice);
    
    // Store reward signal
    safeCudaMemcpy(d_reward_buffer, &reward_signal, 1, hipMemcpyHostToDevice);
    
    // Calculate simulation steps
    int simulation_steps = static_cast<int>(g_config.simulation_time / g_config.dt);
    
    // Reset spike flags
    dim3 block = getOptimalBlockSize();
    dim3 grid = getOptimalGridSize(total_neurons);
    resetSpikeFlags<<<grid, block>>>(d_neurons, total_neurons);
    CUDA_CHECK_KERNEL();
    
    // Simulation loop
    for (int step = 0; step < simulation_steps; ++step) {
        current_time += g_config.dt;
        
        // Inject input current (every few steps to maintain stimulation)
        if (step % 5 == 0) {
            dim3 input_grid = getOptimalGridSize(g_config.input_size);
            injectInputCurrentImproved<<<input_grid, block>>>(
                d_neurons + input_start, d_input_buffer, g_config.input_size, 
                current_time, g_config.input_current_scale
            );
            CUDA_CHECK_KERNEL();
        }
        
        // Update neuron dynamics with RK4
        launchRK4NeuronUpdateKernel(d_neurons, total_neurons, g_config.dt);
        CUDA_CHECK_KERNEL();
        
        // Detect spikes and update spike flags
        safeCudaMemset(d_spike_count, 0, 1);
        launchSpikeDetectionKernel(d_neurons, d_spike_events, g_config.spike_threshold,
                                   d_spike_count, total_neurons, current_time);
        CUDA_CHECK_KERNEL();
        
        // Propagate spikes through synapses
        if (total_synapses > 0) {
            launchSynapseInputKernel(d_synapses, d_neurons, total_synapses);
            CUDA_CHECK_KERNEL();
        }
        
        // Apply reward modulation periodically
        if (step % 10 == 0) {
            applyRewardModulationImproved<<<grid, block>>>(
                d_neurons, total_neurons, reward_signal
            );
            CUDA_CHECK_KERNEL();
        }
    }
    
    // Extract output with improved encoding
    std::vector<float> raw_output(g_config.output_size);
    dim3 output_grid = getOptimalGridSize(g_config.output_size);
    extractOutputImproved<<<output_grid, block>>>(
        d_neurons + output_start, d_output_buffer, g_config.output_size, current_time
    );
    CUDA_CHECK_KERNEL();
    
    safeCudaMemcpy(raw_output.data(), d_output_buffer, g_config.output_size, hipMemcpyDeviceToHost);
    
    // Apply softmax for decision probabilities
    return applySoftmax(raw_output);
}

// Improved softmax with numerical stability
std::vector<float> applySoftmax(const std::vector<float>& input) {
    std::vector<float> output(input.size());
    float max_val = *std::max_element(input.begin(), input.end());
    float sum_exp = 0.0f;
    
    for (size_t i = 0; i < input.size(); ++i) {
        output[i] = expf(input[i] - max_val);
        sum_exp += output[i];
    }
    
    if (sum_exp > 1e-10f) {
        for (size_t i = 0; i < input.size(); ++i) {
            output[i] /= sum_exp;
        }
    } else {
        // Fallback to uniform distribution
        float uniform_prob = 1.0f / input.size();
        std::fill(output.begin(), output.end(), uniform_prob);
    }
    
    return output;
}

// Enhanced STDP with better reward modulation
void updateSynapticWeightsCUDA(float reward_signal) {
    if (total_synapses == 0) return;
    
    // Adaptive STDP parameters based on reward
    float reward_factor = 1.0f + g_config.reward_learning_rate * reward_signal;
    float A_plus = g_config.A_plus * reward_factor;
    float A_minus = g_config.A_minus * (2.0f - reward_factor); // Inverse for depression
    
    // Apply STDP with reward modulation
    launchSTDPUpdateKernel(d_synapses, d_neurons, total_synapses,
                           A_plus, A_minus, g_config.tau_plus, g_config.tau_minus,
                           current_time, g_config.min_weight, g_config.max_weight, 
                           reward_signal);
    CUDA_CHECK_KERNEL();
    
    // Homeostatic mechanisms every 100 updates
    static int update_counter = 0;
    if (++update_counter % 100 == 0 && g_config.homeostatic_strength > 0) {
        applyHomeostaticScaling();
    }
    
    // Update statistics for monitoring
    if (g_config.enable_monitoring && update_counter % g_config.monitoring_interval == 0) {
        updateNetworkStatistics();
    }
    
    g_stats.update_count = update_counter;
    g_stats.reward_signal = reward_signal;
}

// Network statistics computation
void updateNetworkStatistics() {
    // This would compute various network statistics
    // Implementation depends on specific monitoring needs
    static float stats_buffer[4] = {0.0f};
    
    dim3 block = getOptimalBlockSize();
    dim3 grid = getOptimalGridSize(total_neurons);
    computeNetworkStatistics<<<grid, block>>>(
        d_neurons, d_synapses, total_neurons, total_synapses, stats_buffer
    );
    CUDA_CHECK_KERNEL();
    
    // Update global statistics (simplified)
    if (g_stats.update_count % g_config.monitoring_interval == 0) {
        std::cout << "[STATS] Spikes: " << g_stats.total_spikes 
                  << ", Avg Weight: " << g_stats.avg_weight
                  << ", Reward: " << g_stats.reward_signal << std::endl;
    }
}

// Homeostatic scaling to prevent runaway dynamics
void applyHomeostaticScaling() {
    dim3 block = getOptimalBlockSize();
    dim3 grid = getOptimalGridSize(total_synapses);
    
    // Simple homeostatic scaling kernel (would need implementation)
    // This maintains network stability over long training periods
}

// Enhanced cleanup with proper error checking
void cleanupNetwork() {
    std::cout << "[CUDA] Cleaning up network resources..." << std::endl;
    
    if (d_neurons) { hipFree(d_neurons); d_neurons = nullptr; }
    if (d_synapses) { hipFree(d_synapses); d_synapses = nullptr; }
    if (d_input_buffer) { hipFree(d_input_buffer); d_input_buffer = nullptr; }
    if (d_output_buffer) { hipFree(d_output_buffer); d_output_buffer = nullptr; }
    if (d_reward_buffer) { hipFree(d_reward_buffer); d_reward_buffer = nullptr; }
    if (d_spike_events) { hipFree(d_spike_events); d_spike_events = nullptr; }
    if (d_spike_count) { hipFree(d_spike_count); d_spike_count = nullptr; }
    if (d_rng_states) { hipFree(d_rng_states); d_rng_states = nullptr; }
    
    CUDA_CHECK(hipDeviceReset());
    std::cout << "[CUDA] Cleanup complete!" << std::endl;
}

// CUDA Kernel Implementations
__global__ void injectInputCurrentImproved(GPUNeuronState* neurons, const float* input_data, 
                                          int input_size, float current_time, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= input_size) return;
    
    // Enhanced input encoding with noise and scaling
    float normalized_input = tanhf(input_data[idx]); // Normalize to [-1, 1]
    float current = normalized_input * scale;
    
    // Add small amount of noise for better dynamics
    // Note: This is simplified; proper implementation would use hiprand
    float noise = (threadIdx.x % 7 - 3) * 0.5f;
    current += noise;
    
    // Inject current by modifying voltage
    neurons[idx].voltage += current * 0.001f; // Small integration step
    
    // Keep voltage in reasonable range
    neurons[idx].voltage = fminf(fmaxf(neurons[idx].voltage, -85.0f), -35.0f);
}

__global__ void extractOutputImproved(const GPUNeuronState* neurons, float* output_buffer,
                                     int output_size, float current_time) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= output_size) return;
    
    const GPUNeuronState& neuron = neurons[idx];
    
    // Enhanced output encoding combining voltage and spike history
    float voltage_contribution = 1.0f / (1.0f + expf(-(neuron.voltage + 55.0f) / 10.0f));
    
    // Recent spike contribution with exponential decay
    float spike_contribution = 0.0f;
    float time_since_spike = current_time - neuron.last_spike_time;
    if (time_since_spike < 100.0f && time_since_spike >= 0.0f) {
        spike_contribution = expf(-time_since_spike / 30.0f) * 0.8f;
    }
    
    // Combine contributions
    output_buffer[idx] = voltage_contribution + spike_contribution;
    
    // Ensure output is positive
    output_buffer[idx] = fmaxf(output_buffer[idx], 0.0f);
}

__global__ void applyRewardModulationImproved(GPUNeuronState* neurons, int num_neurons, float reward) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_neurons) return;
    
    // Enhanced reward modulation affecting multiple neuron properties
    float modulation = reward * 0.05f; // Scale reward signal
    
    // Modulate leak current (affects excitability)
    neurons[idx].I_leak[0] += modulation;
    neurons[idx].I_leak[0] = fminf(fmaxf(neurons[idx].I_leak[0], -3.0f), 3.0f);
    
    // Slight modulation of gating variables for long-term effects
    if (reward > 0.1f) {
        neurons[idx].m = fminf(neurons[idx].m + modulation * 0.001f, 1.0f);
    } else if (reward < -0.1f) {
        neurons[idx].h = fminf(neurons[idx].h - modulation * 0.001f, 1.0f);
    }
}

__global__ void computeNetworkStatistics(const GPUNeuronState* neurons, const GPUSynapse* synapses,
                                        int num_neurons, int num_synapses, float* stats) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Simplified statistics computation
    if (idx < num_neurons) {
        if (neurons[idx].spiked) {
            atomicAdd(&stats[0], 1.0f); // Spike count
        }
        atomicAdd(&stats[1], neurons[idx].voltage); // Average voltage
    }
    
    if (idx < num_synapses) {
        atomicAdd(&stats[2], fabsf(synapses[idx].weight)); // Average absolute weight
        atomicAdd(&stats[3], synapses[idx].activity_metric); // Activity metric
    }
}

__global__ void resetSpikeFlags(GPUNeuronState* neurons, int num_neurons) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_neurons) {
        neurons[idx].spiked = false;
    }
}