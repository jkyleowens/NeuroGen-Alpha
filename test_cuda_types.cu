// Test file to check CUDA type compatibility
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

__global__ void testKernel() {
    dim3 block(256);
    dim3 grid(1);
    float4 f4 = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    int4 i4 = make_int4(1, 2, 3, 4);
    hipError_t err = hipSuccess;
}

int main() {
    testKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
